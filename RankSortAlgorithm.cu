#include "hip/hip_runtime.h"
// please be noted that This programm will not work without nvidia cuda toolKit, or with google colab
// Rank algorithm but in parallel using cuda instructions
// the function is in the device (GPU), but the main is in the host (CPU) 
%%cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <random>

__global__ void RankSortAlgorithm(int* in_arr, int* out_arr, int size){
    
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int rank = 0;
  if (idx < size) { 
         
    // Count the rank of each index 
    for (int i = 0; i < size; i++) {
    
      if (in_arr[idx] > in_arr[i])
        
        rank++; 
      
        if(in_arr[idx] == in_arr[i] && i < idx)  
          
          rank++;
    }
    
    // copy the results to the output array 
    out_arr[rank] = in_arr[idx]; 
  } 
} 

int main()
{
  int *arr, *sorted_arr;       // initate host copies of arrays
  int *d_arr , *d_sorted_arr; // initate device copies of arrays
  int n = 20;                // predefined size of the array
  int size = sizeof(int) * n;

  // allocate memory in host
  arr = (int*) malloc(size);
  sorted_arr = (int*) malloc(size);
    
 
  printf("The original array : {");
 
  // Random variable assigned for the array  
  for (int i = 0; i < n; i++) {
    arr[i] = rand()%15; 
    printf(" %d ,", arr[i]);
  }
  printf("}\n");

  // allocate memory in device
  hipMalloc((void**) &d_arr, size);
  hipMalloc((void**) &d_sorted_arr, size);
 
  // copy to device
  hipMemcpy(d_arr , arr , size , hipMemcpyHostToDevice);
 
  // call the kernel
  RankSortAlgorithm <<< n , n >>> (d_arr , d_sorted_arr , n);

  // copy back to the host
  hipMemcpy(sorted_arr , d_sorted_arr , size, hipMemcpyDeviceToHost);


  printf("The sorted array : {");
  
  for ( int i = 0; i < n; i++) {
    printf(" %d ,",sorted_arr[i]);
  }
  printf("}");

  // free memory
  free(arr);
  free(sorted_arr);
  hipFree(d_sorted_arr);
  hipFree(d_arr);

return 0;
}